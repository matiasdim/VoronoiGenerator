
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "math.h"

#define THREADS_PER_BLOCK 1024

__global__ void cu_voronoi(int *matrix_d, int *seeds_d, int numberOfSeeds, int width, int sizeOfArray) {
 int x = blockIdx.x * blockDim.x + threadIdx.x;
 int i;
 if (x < sizeOfArray && matrix_d[x] != -1) {
     double minDistance = INFINITY;
     double distance;
     int currentSeed;
     int matrixI = x/width;
     int matrixJ = x % width;
     for (i = 0; i < numberOfSeeds; i++) {
        int candidateSeed = seeds_d[i];
        int seedI = candidateSeed/width;
        int seedJ = candidateSeed % width;
        double xDistance = matrixI - seedI;
        double yDistance = matrixJ - seedJ;
        double distanceSum = pow(xDistance, 2)  + pow(yDistance, 2);
        distance = sqrt(distanceSum);
        if (distance < minDistance) {
            minDistance = distance;
            currentSeed = i;
        }
     }
    matrix_d[x] = currentSeed;
 }
}

extern "C" void voronoi(int *matrix, int *seeds, int numberOfSeeds, int width, int height) {
    int *matrix_d;
    int *seeds_d;
    int sizeOfArray = width * height;
    hipMalloc((void**) &matrix_d, sizeof(int) * sizeOfArray);
    hipMalloc((void**) &seeds_d, sizeof(int) * numberOfSeeds);


    hipMemcpy(matrix_d, matrix, sizeof(int) * sizeOfArray, hipMemcpyHostToDevice);
    hipMemcpy(seeds_d, seeds, sizeof(int) * numberOfSeeds, hipMemcpyHostToDevice);
    int blocks = ceil((float)sizeOfArray/THREADS_PER_BLOCK);
    cu_voronoi <<<blocks, THREADS_PER_BLOCK>>> (matrix_d, seeds_d, numberOfSeeds, width, sizeOfArray);

    hipMemcpy(matrix, matrix_d, sizeof(int) * sizeOfArray, hipMemcpyDeviceToHost);

    hipFree(matrix_d);
    hipFree(seeds_d);
}
